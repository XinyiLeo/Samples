#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#ifndef M_PI
	#define M_PI 3.1415926535897932384626433832795
#endif

const int threadsPerBlock = 256;

__device__ static int dev_N;
__device__ static double dev_omega;
__device__ static int dev_gridN;
__device__ static double dev_dx;
texture<int2, hipTextureType1D, hipReadModeElementType> tex_u;
texture<int2, hipTextureType1D, hipReadModeElementType> tex_g;

__global__ void diff(double* u, double* du, int* row);
__global__ void diffb(double* u, double* du, int* row);

// Sorry for no time to add comment, the update is a combination of red black and rows

int main(int argc, char* argv[]) {
	clock_t start = clock();
	int N = atoi(argv[1]);
	double omega = atof(argv[2]);
	double tol = atof(argv[3]);
	int gridN = N * (2*N-1);
  	const int blocksPerGrid = gridN/threadsPerBlock + (gridN %threadsPerBlock > 0 ? 1 : 0);
  	double* u = (double*)malloc(gridN*sizeof(double));
	double* g = (double*)malloc(gridN*sizeof(double));
  	double* du = (double*)malloc(gridN*sizeof(double));
	double lambda = 100.0;
	double dx = 2.0 / (N-1);
	double maxresid = 0;
	int row = 6;

  	double* dev_u;
	double* dev_g;
  	double* dev_du;
	int* dev_row;

  	hipMalloc((void**)&dev_u, gridN*sizeof(double));
	hipMalloc((void**)&dev_g, gridN*sizeof(double));
  	hipMalloc((void**)&dev_du, gridN*sizeof(double));
	hipMalloc((void**)&dev_row, sizeof(int));

	// Initialize grid
  	for (int x=0; x<2*N-1; x++) {
		for (int y=0; y<N; y++) {
    			u[x * N + y] = 0;
			g[x * N + y] = 10.0 * lambda / sqrt(M_PI) * (exp(-lambda * lambda * ((-3+dx*x)*(-3+dx*x)+(-1+dx*y)*(-1+dx*y))) \
				- exp(-lambda * lambda * ((-1+dx*x)*(-1+dx*x)+(-1+dx*y)*(-1+dx*y))));  
		}
	}

	// Start Loop
	int iter = 0;
	while(iter < 1000) {
		for(int i = 0; i < gridN; i++) {
			du[i] = 0;
		}
		for(row = 0; row < 2*N-1; row ++) {
			hipMemcpyToSymbol(HIP_SYMBOL(dev_N), &N, sizeof(int));
		  	hipMemcpyToSymbol(HIP_SYMBOL(dev_gridN), &gridN, sizeof(int));
			hipMemcpyToSymbol(HIP_SYMBOL(dev_omega), &omega, sizeof(double));
		  	hipMemcpyToSymbol(HIP_SYMBOL(dev_dx), &dx, sizeof(double));
			hipMemcpy(dev_row, &row, sizeof(int), hipMemcpyHostToDevice);
		  	hipMemcpy(dev_u, u, gridN*sizeof(double), hipMemcpyHostToDevice);
			hipMemcpy(dev_g, g, gridN*sizeof(double), hipMemcpyHostToDevice);

		  	hipBindTexture(NULL, tex_u, dev_u, gridN*sizeof(double));
			hipBindTexture(NULL, tex_g, dev_g, gridN*sizeof(double));

		  	diff<<<blocksPerGrid, threadsPerBlock>>>(dev_u, dev_du, dev_row);

		  	hipMemcpy(u, dev_u, gridN*sizeof(double), hipMemcpyDeviceToHost);
			hipMemcpy(du, dev_du, gridN*sizeof(double), hipMemcpyDeviceToHost);
			hipUnbindTexture(tex_u);
			hipUnbindTexture(tex_g);

			hipMemcpyToSymbol(HIP_SYMBOL(dev_N), &N, sizeof(int));
		  	hipMemcpyToSymbol(HIP_SYMBOL(dev_gridN), &gridN, sizeof(int));
			hipMemcpyToSymbol(HIP_SYMBOL(dev_omega), &omega, sizeof(double));
		  	hipMemcpyToSymbol(HIP_SYMBOL(dev_dx), &dx, sizeof(double));
			hipMemcpy(dev_row, &row, sizeof(int), hipMemcpyHostToDevice);
		  	hipMemcpy(dev_u, u, gridN*sizeof(double), hipMemcpyHostToDevice);
			hipMemcpy(dev_g, g, gridN*sizeof(double), hipMemcpyHostToDevice);

		  	hipBindTexture(NULL, tex_u, dev_u, gridN*sizeof(double));
			hipBindTexture(NULL, tex_g, dev_g, gridN*sizeof(double));

		  	diffb<<<blocksPerGrid, threadsPerBlock>>>(dev_u, dev_du, dev_row);

		  	hipMemcpy(u, dev_u, gridN*sizeof(double), hipMemcpyDeviceToHost);
			hipMemcpy(du, dev_du, gridN*sizeof(double), hipMemcpyDeviceToHost);
			hipUnbindTexture(tex_u);
			hipUnbindTexture(tex_g);
		}
		
		maxresid = 0;
		for(int i = 0; i < gridN; i++) { 
	 		maxresid = maxresid > du[i] ? maxresid : du[i];
		}
		if(maxresid < tol)
			break;
		iter ++;
	}
	printf("The final round is: %d", iter);
	// End

  	hipFree(dev_u);
	hipFree(dev_g);
  	hipFree(dev_du);
  	free(u);
	free(g);
  	free(du);
	printf("Elapsed time: %g\n", (float) (clock()-start)/CLOCKS_PER_SEC);
  	return 0;
}

__global__ void diff(double* u, double* du, int* row) {
	// Note up is uu
  	int g_i = (threadIdx.x + blockIdx.x * blockDim.x) % dev_gridN;
		int g_y = g_i % dev_N;
		int g_x = (g_i - g_y) / dev_N;
	  	int g_iu = g_i-dev_N;
	  	int g_id = g_i+dev_N;
	  	int g_il = g_i-1;
	  	int g_ir = g_i+1;
		double resid;
		int2 m_int2 = tex1Dfetch(tex_u, g_i);
	  	double m = __hiloint2double(m_int2.y, m_int2.x);
		int2 gm_int2 = tex1Dfetch(tex_g, g_i);	
		double gm = __hiloint2double(gm_int2.y, gm_int2.x);
		if(g_x == *row && g_i % 2==0) {
			if(g_y != 0 and g_y != dev_N-1) {
				int2 r_int2 = tex1Dfetch(tex_u, g_ir);
				int2 l_int2 = tex1Dfetch(tex_u, g_il);
		  		double r = __hiloint2double(r_int2.y, r_int2.x);
		  		double l = __hiloint2double(l_int2.y, l_int2.x);
				if(g_x == 0) {
					int2 d_int2 = tex1Dfetch(tex_u, g_id);
		  			double d = __hiloint2double(d_int2.y, d_int2.x);
					resid = 0.25*(l+r+d-3*m)-dev_dx*dev_dx/4*gm;
					u[g_i] += dev_omega * resid;
				}
				else if(g_x == 2*dev_N-1) {
					int2 uu_int2 = tex1Dfetch(tex_u, g_iu);
		  			double uu = __hiloint2double(uu_int2.y, uu_int2.x);
					resid = 0.25*(l+r+uu-3*m)-dev_dx*dev_dx/4*gm;
					u[g_i] += dev_omega * resid;
				}
				else {
					int2 uu_int2 = tex1Dfetch(tex_u, g_iu);
		  			double uu = __hiloint2double(uu_int2.y, uu_int2.x);
					int2 d_int2 = tex1Dfetch(tex_u, g_id);
		  			double d = __hiloint2double(d_int2.y, d_int2.x);
					resid = 0.25*(l+r+uu+d-4*m)-dev_dx*dev_dx/4*gm;
					u[g_i] += dev_omega * resid;	
				}
				du[g_i] = resid;			
			}
			else {
				du[g_i] = 0;
			}
		}
}


__global__ void diffb(double* u, double* du, int* row) {
	// Note up is uu
  	int g_i = (threadIdx.x + blockIdx.x * blockDim.x) % dev_gridN;
		int g_y = g_i % dev_N;
		int g_x = (g_i - g_y) / dev_N;
	  	int g_iu = g_i-dev_N;
	  	int g_id = g_i+dev_N;
	  	int g_il = g_i-1;
	  	int g_ir = g_i+1;
		double resid;
		int2 m_int2 = tex1Dfetch(tex_u, g_i);
	  	double m = __hiloint2double(m_int2.y, m_int2.x);
		int2 gm_int2 = tex1Dfetch(tex_g, g_i);	
		double gm = __hiloint2double(gm_int2.y, gm_int2.x);
		if(g_x == *row && g_i % 2==1) {
			if(g_y != 0 and g_y != dev_N-1) {
				int2 r_int2 = tex1Dfetch(tex_u, g_ir);
				int2 l_int2 = tex1Dfetch(tex_u, g_il);
		  		double r = __hiloint2double(r_int2.y, r_int2.x);
		  		double l = __hiloint2double(l_int2.y, l_int2.x);
				if(g_x == 0) {
					int2 d_int2 = tex1Dfetch(tex_u, g_id);
		  			double d = __hiloint2double(d_int2.y, d_int2.x);
					resid = 0.25*(l+r+d-3*m)-dev_dx*dev_dx/4*gm;
					u[g_i] += dev_omega * resid;
				}
				else if(g_x == 2*dev_N-1) {
					int2 uu_int2 = tex1Dfetch(tex_u, g_iu);
		  			double uu = __hiloint2double(uu_int2.y, uu_int2.x);
					resid = 0.25*(l+r+uu-3*m)-dev_dx*dev_dx/4*gm;
					u[g_i] += dev_omega * resid;
				}
				else {
					int2 uu_int2 = tex1Dfetch(tex_u, g_iu);
		  			double uu = __hiloint2double(uu_int2.y, uu_int2.x);
					int2 d_int2 = tex1Dfetch(tex_u, g_id);
		  			double d = __hiloint2double(d_int2.y, d_int2.x);
					resid = 0.25*(l+r+uu+d-4*m)-dev_dx*dev_dx/4*gm;
					u[g_i] += dev_omega * resid;	
				}
				du[g_i] = resid;			
			}
			else {
				du[g_i] = 0;
			}
		}
}






