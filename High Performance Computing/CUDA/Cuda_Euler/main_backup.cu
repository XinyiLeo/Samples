#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <magma.h>
#include <magma_types.h>
#include <magma_lapack.h>
#include "hipsparse.h" 

#ifndef M_PI
#define M_PI 3.1415926535897932384626433832795
#endif

/*
	Xinyi Wu

	Start: 2018/6/12	
		
	This is the cuda version of Linear Euler Equation. I will implement the ADI method here. 
	There is only two input N, M. N is the length of the grid, M is the step of the time.
	To make things simple, I will express the grid as 1D array A[x * N + y]. x is the number of column
	y number of row. 

	after set up I will simply loop. 
	use magma for solving linear equations and cusparse for matrix operation
*/


const int threadsPerBlock = 256;

__device__ static int dev_N;
__device__ static double dev_gamma;
__device__ static double dev_dx;
texture<int2, hipTextureType1D, hipReadModeElementType> tex_rho;
texture<int2, hipTextureType1D, hipReadModeElementType> tex_u;
texture<int2, hipTextureType1D, hipReadModeElementType> tex_v;
texture<int2, hipTextureType1D, hipReadModeElementType> tex_p;
texture<int2, hipTextureType1D, hipReadModeElementType> tex_mulD;

__global__ void plusrhox(double* rho);
__global__ void pluspx(double* p);
__global__ void plusux(double* u);
__global__ void plusrhoy(double* rho);
__global__ void pluspy(double* p);
__global__ void plusvy(double* v);


int main(int argc, char* argv[]) {
	// Initialization: Basic Parameters
	int N = atoi(argv[1]);
	int M = atoi(argv[2]);
	double gamma = 1.4;
	double dx = 2.0 / N;
	double dt = 2.0 / M;
	double dd = dt / (4*dx);
	int baseidx = 0;
	
	const int blocksPerGrid = N/threadsPerBlock + (N%threadsPerBlock > 0 ? 1 : 0);

	// Initialization: Grids
	double* rho = (double*)malloc(N*N*sizeof(double));
	double* u = (double*)malloc(N*N*sizeof(double));
	double* v = (double*)malloc(N*N*sizeof(double));
	double* p = (double*)malloc(N*N*sizeof(double));
	double* mulD = (double*)malloc(N*N*sizeof(double)); // The temporary result after a matrix multiplies D

	double* dev_rho;
	double* dev_u;
	double* dev_v;
	double* dev_p;
	double* dev_mulD;

  	double* Aval = (double*)malloc(2*N*sizeof(double));  // MulD Prepare
  	int* Arowptr = (int*)malloc((N+1)*sizeof(int));
  	int* Acolind = (int*)malloc(2*N*sizeof(int));
  	double* xval = (double*)malloc(N*sizeof(double));
  	int* xind = (int*)malloc(N*sizeof(int));
  	double* y = (double*)malloc(N*sizeof(double));
	
	double* dev_Aval;
  	int* dev_Arowptr;
  	int* dev_Acolind;
  	double* dev_xval;
  	int* dev_xind;
  	double* dev_x;
  	double* dev_y;

  	hipMalloc((void**)&dev_Aval, 2*N*sizeof(double));
  	hipMalloc((void**)&dev_Arowptr, (N+1)*sizeof(int));
  	hipMalloc((void**)&dev_Acolind, 2*N*sizeof(int));
  	hipMalloc((void**)&dev_xval, N*sizeof(double));
  	hipMalloc((void**)&dev_xind, N*sizeof(int));
  	hipMalloc((void**)&dev_x, N*sizeof(double));
  	hipMalloc((void**)&dev_y, N*sizeof(double));

	hipMalloc((void**)&dev_rho, N*N*sizeof(double));
	hipMalloc((void**)&dev_u, N*N*sizeof(double));
	hipMalloc((void**)&dev_v, N*N*sizeof(double));
	hipMalloc((void**)&dev_p, N*N*sizeof(double));
	hipMalloc((void**)&dev_mulD, N*N*sizeof(double));

	hipBindTexture(NULL, tex_rho, dev_rho, N*N*sizeof(double));
	hipBindTexture(NULL, tex_u, dev_u, N*N*sizeof(double));
	hipBindTexture(NULL, tex_v, dev_v, N*N*sizeof(double));
	hipBindTexture(NULL, tex_p, dev_p, N*N*sizeof(double));
	hipBindTexture(NULL, tex_mulD, dev_mulD, N*N*sizeof(double));

	// Build D
 	for(int i = 0; i < N; i++) {
		Aval[2*i] = -dd;
		Aval[2*i+1] = dd;
		Arowptr[i] = 2*i;
		Acolind[2*i] = i-1;
		Acolind[2*i+1] = i+1;
	}
	Acolind[0] = 1;
	Acolind[1] = N-1;
	Acolind[2*N-2] = 0;
	Acolind[2*N-1] = N-2;
	Arowptr[N] = 2*N;
	Aval[0] = dd;
	Aval[1] = -dd;
	Aval[2*N-2] = dd;
	Aval[2*N-1] = -dd;
  	int nnz = 2 * N;

	// Initialize the grid
	for (int x = 0; x < N; x++) {
		for (int y = 0; y < N; y++) {
			rho[x*N+y] = 2.0 / gamma * exp(-100*((-1+x*dx)*(-1+x*dx)+(-1+y*dx)*(-1+y*dx)));
			u[x*N+y] = 0;
			v[x*N+y] = 0;
			p[x*N+y] = 2.0 * exp(-100*((-1+x*dx)*(-1+x*dx)+(-1+y*dx)*(-1+y*dx)));
		}
	}

	// Start For loop
	for(int loop = 0; loop < 1; loop++) {
		// Explicit in x
		// Updating rho
		// u * D
		for(int k = 0; k < N; k++) {
			for(int i = 0; i < N; i++) {
				xind[i] = i;
				xval[i] = u[k * N + i];
			}

		  	hipMemcpy(dev_Aval, Aval, 2*N*sizeof(double), hipMemcpyHostToDevice);
		  	hipMemcpy(dev_Arowptr, Arowptr, (N+1)*sizeof(int), hipMemcpyHostToDevice);
		  	hipMemcpy(dev_Acolind, Acolind, 2*N*sizeof(int), hipMemcpyHostToDevice);
		  	hipMemcpy(dev_xval, xval, N*sizeof(double), hipMemcpyHostToDevice);
		  	hipMemcpy(dev_xind, xind, N*sizeof(int), hipMemcpyHostToDevice);

		  	hipsparseHandle_t handle = NULL;
		  	hipsparseCreate(&handle);
		  	hipsparseMatDescr_t descr = NULL; 
		  	hipsparseCreateMatDescr(&descr);
		  	hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
		  	hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);
		  	hipsparseDsctr(handle, N, dev_xval, dev_xind, dev_x, HIPSPARSE_INDEX_BASE_ZERO);
		  	double one = 1.;
		  	double zero = 0.;
		  	hipsparseDcsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, nnz, &one, descr, \
				dev_Aval, dev_Arowptr, dev_Acolind, dev_x, &zero, dev_y);
		  	hipDeviceSynchronize();
		  	hipMemcpy(y, dev_y, N*sizeof(double), hipMemcpyDeviceToHost);

			for(int i = 0; i < N; i++) {
				 mulD[k * N + i] = y[i];
			}
		}
		
		// Rho linear algebra update
		hipMemcpyToSymbol(HIP_SYMBOL(dev_N), &N, sizeof(int));
		hipMemcpyToSymbol(HIP_SYMBOL(dev_dx), &dx, sizeof(double));
		hipMemcpy(dev_rho, rho, N*N*sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(dev_mulD, mulD, N*N*sizeof(double), hipMemcpyHostToDevice);
		plusrhox<<<blocksPerGrid, threadsPerBlock>>>(dev_rho);
		hipMemcpy(rho, dev_rho, N*N*sizeof(double), hipMemcpyDeviceToHost);


		// Updating p
		// p linear algebra update
		hipMemcpyToSymbol(HIP_SYMBOL(dev_N), &N, sizeof(int));
		hipMemcpyToSymbol(HIP_SYMBOL(dev_dx), &dx, sizeof(double));
		hipMemcpyToSymbol(HIP_SYMBOL(dev_gamma), &gamma, sizeof(double));
		hipMemcpy(dev_p, p, N*N*sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(dev_mulD, mulD, N*N*sizeof(double), hipMemcpyHostToDevice);
		pluspx<<<blocksPerGrid, threadsPerBlock>>>(dev_p);
		hipMemcpy(p, dev_p, N*N*sizeof(double), hipMemcpyDeviceToHost);
		
		// Updating u
		// p * D
		for(int k = 0; k < N; k++) {
			for(int i = 0; i < N; i++) {
				xind[i] = i;
				xval[i] = p[k * N + i];
			}

		  	hipMemcpy(dev_Aval, Aval, 2*N*sizeof(double), hipMemcpyHostToDevice);
		  	hipMemcpy(dev_Arowptr, Arowptr, (N+1)*sizeof(int), hipMemcpyHostToDevice);
		  	hipMemcpy(dev_Acolind, Acolind, 2*N*sizeof(int), hipMemcpyHostToDevice);
		  	hipMemcpy(dev_xval, xval, N*sizeof(double), hipMemcpyHostToDevice);
		  	hipMemcpy(dev_xind, xind, N*sizeof(int), hipMemcpyHostToDevice);

		  	hipsparseHandle_t handle = NULL;
		  	hipsparseCreate(&handle);
		  	hipsparseMatDescr_t descr = NULL; 
		  	hipsparseCreateMatDescr(&descr);
		  	hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
		  	hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);
		  	hipsparseDsctr(handle, N, dev_xval, dev_xind, dev_x, HIPSPARSE_INDEX_BASE_ZERO);
		  	double one = 1.;
		  	double zero = 0.;
		  	hipsparseDcsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, nnz, &one, descr, \
				dev_Aval, dev_Arowptr, dev_Acolind, dev_x, &zero, dev_y);
		  	hipDeviceSynchronize();
		  	hipMemcpy(y, dev_y, N*sizeof(double), hipMemcpyDeviceToHost);

			for(int i = 0; i < N; i++) {
				 mulD[k * N + i] = y[i];
			}
		}
		// u linear algebra update
		hipMemcpyToSymbol(HIP_SYMBOL(dev_N), &N, sizeof(int));
		hipMemcpyToSymbol(HIP_SYMBOL(dev_dx), &dx, sizeof(double));
		hipMemcpyToSymbol(HIP_SYMBOL(dev_gamma), &gamma, sizeof(double));
		hipMemcpy(dev_u, u, N*N*sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(dev_mulD, mulD, N*N*sizeof(double), hipMemcpyHostToDevice);
		plusux<<<blocksPerGrid, threadsPerBlock>>>(dev_u);
		hipMemcpy(u, dev_u, N*N*sizeof(double), hipMemcpyDeviceToHost);

		// Implicit in y
  		magma_init();
  		magma_int_t *piv, info;
  		double* A;
  		double* b;
  		magma_dmalloc_pinned(&A, 16*N*N);
  		magma_dmalloc_pinned(&b, 4*N*N);
  		piv = (magma_int_t*)malloc(4*N*sizeof(magma_int_t));

	  	for(int i = 0; i < 4*N; i++) {
			A[(4*N+1)*i] = 1.0;
		}
		baseidx = 8*N*N;
		for(int i = 0; i < N; i++) {
			A[baseidx + i*4*N + (i+1)%N] = -dd;
			A[baseidx + i*4*N + (i-1+N)%N] = dd;
		}
		baseidx = 8*N*N + 3*N;
		for(int i = 0; i < N; i++) {
			A[baseidx + i*4*N + (i+1)%N] = -gamma*dd;
			A[baseidx + i*4*N + (i-1+N)%N] = gamma*dd;
		}
		baseidx = 12*N*N + 2*N;
		for(int i = 0; i < N; i++) {
			A[baseidx + i*4*N + (i+1)%N] = -dd;
			A[baseidx + i*4*N + (i-1+N)%N] = dd;
		}

		for(int x = 0; x < N; x++) {
			for(int y = 0; y < N; y++) {
				b[x*4*N+y] = rho[x*N+y];
				b[x*4*N+N+y] = u[x*N+y];
				b[x*4*N+2*N+y] = v[x*N+y];
				b[x*4*N+3*N+y] = p[x*N+y];
			}
		}
  		magma_dgesv(4*N, N, A, 4*N, piv, b, 4*N, &info);
		for(int x = 0; x < N; x++) {
			for(int y = 0; y < N; y++) {
				rho[x*N+y] = b[x*4*N+y];
				u[x*N+y] = b[x*4*N+N+y];
				v[x*N+y] = b[x*4*N+2*N+y];
				p[x*N+y] = b[x*4*N+3*N+y];
			}
		}
		
		// Explicit in y
		// Updating rho
		// v * D
		for(int k = 0; k < N; k++) {
			for(int i = 0; i < N; i++) {
				xind[i] = i;
				xval[i] = v[k * N + i];
			}

		  	hipMemcpy(dev_Aval, Aval, 2*N*sizeof(double), hipMemcpyHostToDevice);
		  	hipMemcpy(dev_Arowptr, Arowptr, (N+1)*sizeof(int), hipMemcpyHostToDevice);
		  	hipMemcpy(dev_Acolind, Acolind, 2*N*sizeof(int), hipMemcpyHostToDevice);
		  	hipMemcpy(dev_xval, xval, N*sizeof(double), hipMemcpyHostToDevice);
		  	hipMemcpy(dev_xind, xind, N*sizeof(int), hipMemcpyHostToDevice);

		  	hipsparseHandle_t handle = NULL;
		  	hipsparseCreate(&handle);
		  	hipsparseMatDescr_t descr = NULL; 
		  	hipsparseCreateMatDescr(&descr);
		  	hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
		  	hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);
		  	hipsparseDsctr(handle, N, dev_xval, dev_xind, dev_x, HIPSPARSE_INDEX_BASE_ZERO);
		  	double one = 1.;
		  	double zero = 0.;
		  	hipsparseDcsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, nnz, &one, descr, \
				dev_Aval, dev_Arowptr, dev_Acolind, dev_x, &zero, dev_y);
		  	hipDeviceSynchronize();
		  	hipMemcpy(y, dev_y, N*sizeof(double), hipMemcpyDeviceToHost);

			for(int i = 0; i < N; i++) {
				 mulD[k * N + i] = y[i];
			}
		}
		
		// Rho linear algebra update
		hipMemcpyToSymbol(HIP_SYMBOL(dev_N), &N, sizeof(int));
		hipMemcpyToSymbol(HIP_SYMBOL(dev_dx), &dx, sizeof(double));
		hipMemcpy(dev_rho, rho, N*N*sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(dev_mulD, mulD, N*N*sizeof(double), hipMemcpyHostToDevice);
		plusrhoy<<<blocksPerGrid, threadsPerBlock>>>(dev_rho);
		hipMemcpy(rho, dev_rho, N*N*sizeof(double), hipMemcpyDeviceToHost);

		// Updating p
		// p linear algebra update
		hipMemcpyToSymbol(HIP_SYMBOL(dev_N), &N, sizeof(int));
		hipMemcpyToSymbol(HIP_SYMBOL(dev_dx), &dx, sizeof(double));
		hipMemcpyToSymbol(HIP_SYMBOL(dev_gamma), &gamma, sizeof(double));
		hipMemcpy(dev_p, p, N*N*sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(dev_mulD, mulD, N*N*sizeof(double), hipMemcpyHostToDevice);
		pluspy<<<blocksPerGrid, threadsPerBlock>>>(dev_p);
		hipMemcpy(p, dev_p, N*N*sizeof(double), hipMemcpyDeviceToHost);
	
		// Updating v
		// p * D
		for(int k = 0; k < N; k++) {
			for(int i = 0; i < N; i++) {
				xind[i] = i;
				xval[i] = p[k * N + i];
			}

		  	hipMemcpy(dev_Aval, Aval, 2*N*sizeof(double), hipMemcpyHostToDevice);
		  	hipMemcpy(dev_Arowptr, Arowptr, (N+1)*sizeof(int), hipMemcpyHostToDevice);
		  	hipMemcpy(dev_Acolind, Acolind, 2*N*sizeof(int), hipMemcpyHostToDevice);
		  	hipMemcpy(dev_xval, xval, N*sizeof(double), hipMemcpyHostToDevice);
		  	hipMemcpy(dev_xind, xind, N*sizeof(int), hipMemcpyHostToDevice);

		  	hipsparseHandle_t handle = NULL;
		  	hipsparseCreate(&handle);
		  	hipsparseMatDescr_t descr = NULL; 
		  	hipsparseCreateMatDescr(&descr);
		  	hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
		  	hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);
		  	hipsparseDsctr(handle, N, dev_xval, dev_xind, dev_x, HIPSPARSE_INDEX_BASE_ZERO);
		  	double one = 1.;
		  	double zero = 0.;
		  	hipsparseDcsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, nnz, &one, descr, \
				dev_Aval, dev_Arowptr, dev_Acolind, dev_x, &zero, dev_y);
		  	hipDeviceSynchronize();
		  	hipMemcpy(y, dev_y, N*sizeof(double), hipMemcpyDeviceToHost);

			for(int i = 0; i < N; i++) {
				 mulD[k * N + i] = y[i];
			}
		}
		// v linear algebra update
		hipMemcpyToSymbol(HIP_SYMBOL(dev_N), &N, sizeof(int));
		hipMemcpyToSymbol(HIP_SYMBOL(dev_dx), &dx, sizeof(double));
		hipMemcpyToSymbol(HIP_SYMBOL(dev_gamma), &gamma, sizeof(double));
		hipMemcpy(dev_v, v, N*N*sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(dev_mulD, mulD, N*N*sizeof(double), hipMemcpyHostToDevice);
		plusvy<<<blocksPerGrid, threadsPerBlock>>>(dev_v);
		hipMemcpy(v, dev_v, N*N*sizeof(double), hipMemcpyDeviceToHost);

		// Implicit in x
  		magma_init();
  		magma_int_t *piv, info;
  		double* A;
  		double* b;
  		magma_dmalloc_pinned(&A, 16*N*N);
  		magma_dmalloc_pinned(&b, 4*N*N);
  		piv = (magma_int_t*)malloc(4*N*sizeof(magma_int_t));

	  	for(int i = 0; i < 4*N; i++) {
			A[(4*N+1)*i] = 1.0;
		}
		baseidx = 4*N*N;
		for(int i = 0; i < N; i++) {
			A[baseidx + i*4*N + (i+1)%N] = -dd;
			A[baseidx + i*4*N + (i-1+N)%N] = dd;
		}
		baseidx = 4*N*N + 3*N;
		for(int i = 0; i < N; i++) {
			A[baseidx + i*4*N + (i+1)%N] = -gamma*dd;
			A[baseidx + i*4*N + (i-1+N)%N] = gamma*dd;
		}
		baseidx = 12*N*N + N;
		for(int i = 0; i < N; i++) {
			A[baseidx + i*4*N + (i+1)%N] = -dd;
			A[baseidx + i*4*N + (i-1+N)%N] = dd;
		}

		for(int x = 0; x < N; x++) {
			for(int y = 0; y < N; y++) {
				b[x*4*N+y] = rho[x*N+y];
				b[x*4*N+N+y] = u[x*N+y];
				b[x*4*N+2*N+y] = v[x*N+y];
				b[x*4*N+3*N+y] = p[x*N+y];
			}
		}
  		magma_dgesv(4*N, N, A, 4*N, piv, b, 4*N, &info);
		for(int x = 0; x < N; x++) {
			for(int y = 0; y < N; y++) {
				rho[x*N+y] = b[x*4*N+y];
				u[x*N+y] = b[x*4*N+N+y];
				v[x*N+y] = b[x*4*N+2*N+y];
				p[x*N+y] = b[x*4*N+3*N+y];
			}
		}
		
  		magma_free_pinned(A);
  		magma_free_pinned(b);
  		free(piv);
  		magma_finalize();
	}

	// End
	hipUnbindTexture(tex_rho);
	hipUnbindTexture(tex_u);
	hipUnbindTexture(tex_v);
	hipUnbindTexture(tex_p);
	hipUnbindTexture(tex_mulD);
	hipFree(dev_rho); 
	hipFree(dev_u); 
	hipFree(dev_v); 
	hipFree(dev_p); 
	hipFree(dev_mulD); 
	free(rho);
	free(u);
	free(v);
	free(p);
  	free(Aval);
  	free(Arowptr);
  	free(Acolind);
  	free(xval);
  	free(xind);
  	free(y);
	free(mulD);
	return 0;
}

__global__ void plusrhox(double* rho) {
  	int g_i = (threadIdx.x + blockIdx.x * blockDim.x) % (dev_N * dev_N);
	int2 rhoval_int2 = tex1Dfetch(tex_rho, g_i);
	int2 mulDval_int2 = tex1Dfetch(tex_mulD, g_i);
  	double rhoval = __hiloint2double(rhoval_int2.y, rhoval_int2.x);
	double mulDval = __hiloint2double(mulDval_int2.y, mulDval_int2.x);
	rho[g_i] -= mulDval; 
}

__global__ void pluspx(double* p) {
  	int g_i = (threadIdx.x + blockIdx.x * blockDim.x) % (dev_N * dev_N);
	int2 pval_int2 = tex1Dfetch(tex_p, g_i);
	int2 mulDval_int2 = tex1Dfetch(tex_mulD, g_i);
  	double pval = __hiloint2double(pval_int2.y, pval_int2.x);
	double mulDval = __hiloint2double(mulDval_int2.y, mulDval_int2.x);
	p[g_i] -= dev_gamma * mulDval; 
}

__global__ void plusux(double* u) {
  	int g_i = (threadIdx.x + blockIdx.x * blockDim.x) % (dev_N * dev_N);
	int2 uval_int2 = tex1Dfetch(tex_u, g_i);
	int2 mulDval_int2 = tex1Dfetch(tex_mulD, g_i);
  	double uval = __hiloint2double(uval_int2.y, uval_int2.x);
	double mulDval = __hiloint2double(mulDval_int2.y, mulDval_int2.x);
	u[g_i] -= mulDval; 
}

__global__ void plusrhoy(double* rho) {
  	int g_i = (threadIdx.x + blockIdx.x * blockDim.x) % (dev_N * dev_N);
	int2 rhoval_int2 = tex1Dfetch(tex_rho, g_i);
	int2 mulDval_int2 = tex1Dfetch(tex_mulD, g_i);
  	double rhoval = __hiloint2double(rhoval_int2.y, rhoval_int2.x);
	double mulDval = __hiloint2double(mulDval_int2.y, mulDval_int2.x);
	rho[g_i] -= mulDval; 
}

__global__ void pluspy(double* p) {
  	int g_i = (threadIdx.x + blockIdx.x * blockDim.x) % (dev_N * dev_N);
	int2 pval_int2 = tex1Dfetch(tex_p, g_i);
	int2 mulDval_int2 = tex1Dfetch(tex_mulD, g_i);
  	double pval = __hiloint2double(pval_int2.y, pval_int2.x);
	double mulDval = __hiloint2double(mulDval_int2.y, mulDval_int2.x);
	p[g_i] -= dev_gamma * mulDval; 
}

__global__ void plusvy(double* v) {
  	int g_i = (threadIdx.x + blockIdx.x * blockDim.x) % (dev_N * dev_N);
	int2 vval_int2 = tex1Dfetch(tex_v, g_i);
	int2 mulDval_int2 = tex1Dfetch(tex_mulD, g_i);
  	double vval = __hiloint2double(vval_int2.y, vval_int2.x);
	double mulDval = __hiloint2double(mulDval_int2.y, mulDval_int2.x);
	v[g_i] -= mulDval; 
}



