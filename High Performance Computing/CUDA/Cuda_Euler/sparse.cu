/*
  Example using cuSPARSE matrix library
  
  Program written by David Chopp
  
  Example of using the cuSPARSE package to do a sparse matrix vector multiplication
  
  Inputs: none
  
  Outputs: Shows the equation that is computed.
  
  Editing History:
  6/4/14: Initial draft
*/
#include <stdio.h>
#include "hipsparse.h" // this is the header file for using the cuSPARSE library

/*
int main(int argc, char* argv[])

The main program takes no arguments and creates a tridiagonal matrix and
then multiplies it by a simple sparse vector.  

Inputs: none

Output: prints the expression computed in matrix format.
*/
int main(int argc, char* argv[]) {

  // Dimensions of the matrix for this example
  const int N = 9;
  
  // The data vector for compressed sparse row format
  double* Aval = (double*)malloc(3*N*sizeof(double));
  
  // The row index and column index vectors for the compressed sparse row format
  int* Arowptr = (int*)malloc((N+1)*sizeof(int));
  int* Acolind = (int*)malloc(3*N*sizeof(int));
  
  // The data vector for a sparse vector
  double* xval = (double*)malloc(N*sizeof(double));
  
  // The index vector for a sparse vector
  int* xind = (int*)malloc(N*sizeof(int));
  
  // A dense solution vector.
  double* y = (double*)malloc(N*sizeof(double));

  // build the matrix A with -2 down the diagonal and ones on the off diagonals
  // in compressed sparse row format
  int row;
  int index = 0;
  for (row=0; row<N; ++row) {
    Arowptr[row] = index;
    if (row > 0) {
      Aval[index] = 1.;
      Acolind[index++] = row-1;
    }
    Aval[index] = -2.;
    Acolind[index++] = row;
    if (row < N-1) {
      Aval[index] = 1.;
      Acolind[index++] = row+1;
    }
  }
  Arowptr[N] = index;

  for(int i = 0; i < 3*N; i++) 
    printf("Aval[%d] = %lf\n", i, Aval[i]);

  for(int i = 0; i < 1+N; i++) 
    printf("Arowptr[%d] = %d\n", i, Arowptr[i]);

  for(int i = 0; i < 3*N; i++)
    printf("Acolind[%d] = %d\n", i, Acolind[i]);


  int nnz = index;

  // build the vector x as a sparse vector with alternating 1 0 2 0 3 ...
  index = 0;
  for (row=0; row<N; row += 2) {
    xval[index] = (row/2)+1;
    xind[index++] = row;
  }
  for(int i = 0; i < 1+N; i++) 
    printf("xval[%d] = %lf\n", i, xval[i]);

  for(int i = 0; i < N; i++)
    printf("xind[%d] = %d\n", i, xind[i]);

  // All operations take place on the device, so must create versions of all the input
  // and output data on the device and copy the input data to the device.
  double* dev_Aval;
  int* dev_Arowptr;
  int* dev_Acolind;
  double* dev_xval;
  int* dev_xind;
  double* dev_x;
  double* dev_y;
  hipMalloc((void**)&dev_Aval, 3*N*sizeof(double));
  hipMalloc((void**)&dev_Arowptr, (N+1)*sizeof(int));
  hipMalloc((void**)&dev_Acolind, 3*N*sizeof(int));
  hipMalloc((void**)&dev_xval, N*sizeof(double));
  hipMalloc((void**)&dev_xind, N*sizeof(int));
  hipMalloc((void**)&dev_x, N*sizeof(double));
  hipMalloc((void**)&dev_y, N*sizeof(double));

  hipMemcpy(dev_Aval, Aval, 3*N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dev_Arowptr, Arowptr, (N+1)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_Acolind, Acolind, 3*N*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_xval, xval, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dev_xind, xind, N*sizeof(int), hipMemcpyHostToDevice);

  // Initialize the cuSPARSE library and get the environment context
  hipsparseHandle_t handle = NULL;
  hipsparseCreate(&handle);

  // Create a matrix description for our sparse matrix
  hipsparseMatDescr_t descr = NULL; 
  hipsparseCreateMatDescr(&descr);
  hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);

  // Convert the sparse vector into a dense vector
  hipsparseDsctr(handle, (N+1)/2, dev_xval, dev_xind, dev_x, HIPSPARSE_INDEX_BASE_ZERO);

  // Multiple the matrix A by the vector x
  double one = 1.;
  double zero = 0.;
  hipsparseDcsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, nnz, &one, descr,
        dev_Aval, dev_Arowptr, dev_Acolind, dev_x, &zero, dev_y);

  // Ensure the operation is complete.
  hipDeviceSynchronize();
  
  // Copy the results on the device back to the host
  hipMemcpy(y, dev_y, N*sizeof(double), hipMemcpyDeviceToHost);

  // Print the results
  int i, j, mindex = 0, vindex = 0;
  for (i=0; i<N; ++i) {
    printf("[ ");
    for (j=0; j<N; ++j) {
      if (mindex < Arowptr[i+1] && j==Acolind[mindex]) {
        printf("%5.1f", Aval[mindex++]);
      } else {
        printf("%5.1f", 0.);
      }
    }
    printf("] [");
    if (i == xind[vindex]) {
      printf("%5.1f]", xval[vindex++]);
    } else {
      printf("%5.1f]", 0.);
    }
    if (i == N/2) {
      printf(" = [");
    } else {
      printf("   [");
    }
    printf("%5.1f]\n", y[i]);
  }

  return 0;
}
